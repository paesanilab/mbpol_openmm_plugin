#include "hip/hip_runtime.h"
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real3 pos;
    real3 field, fieldPolar, inducedDipole, inducedDipolePolar;
    float damp;
    int moleculeIndex;
    int atomType;
} AtomData;

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq, const real* __restrict__ inducedDipole,
        const real* __restrict__ inducedDipolePolar, const float* __restrict__ damping, const int* __restrict__ moleculeIndex, const int* __restrict__ atomType) {
    real4 atomPosq = posq[atom];
    data.pos = make_real3(atomPosq.x, atomPosq.y, atomPosq.z);
    data.inducedDipole.x = inducedDipole[atom*3];
    data.inducedDipole.y = inducedDipole[atom*3+1];
    data.inducedDipole.z = inducedDipole[atom*3+2];
    data.inducedDipolePolar.x = inducedDipolePolar[atom*3];
    data.inducedDipolePolar.y = inducedDipolePolar[atom*3+1];
    data.inducedDipolePolar.z = inducedDipolePolar[atom*3+2];
    data.damp = damping[atom];
    data.moleculeIndex = moleculeIndex[atom];
    data.atomType = atomType[atom];
}

inline __device__ void zeroAtomData(AtomData& data) {
    data.field = make_real3(0);
    data.fieldPolar = make_real3(0);
}

#ifdef USE_EWALD
__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, bool isSelfInteraction) {
    if (isSelfInteraction)
        return;
    real scale1, scale2;
    real r2 = dot(deltaR, deltaR);
    bool isSameWater = atom1.moleculeIndex == atom2.moleculeIndex;
    if (r2 < CUTOFF_SQUARED) {
        real rI = RSQRT(r2);
        real r = RECIP(rI);

        // calculate the error function damping terms

        real ralpha = EWALD_ALPHA*r;
        real exp2a = EXP(-(ralpha*ralpha));
#ifdef USE_DOUBLE_PRECISION
        const real erfcAlphaR = erfc(ralpha);
#else
        // This approximation for erfc is from Abramowitz and Stegun (1964) p. 299.  They cite the following as
        // the original source: C. Hastings, Jr., Approximations for Digital Computers (1955).  It has a maximum
        // error of 1.5e-7.

        const real t = RECIP(1.0f+0.3275911f*ralpha);
        const real erfcAlphaR = (0.254829592f+(-0.284496736f+(1.421413741f+(-1.453152027f+1.061405429f*t)*t)*t)*t)*t*exp2a;
#endif
        // FIXME thole copy in unique location
        const enum TholeIndices { TCC, TCD, TDD, TDDOH, TDDHH };
        const float thole[5] =  { 0.4, 0.4, 0.4,   0.4,   0.4 };
        real bn0 = erfcAlphaR*rI;
        real alsq2 = 2*EWALD_ALPHA*EWALD_ALPHA;
        real alsq2n = RECIP(SQRT_PI*EWALD_ALPHA);
        alsq2n *= alsq2;
        real bn1 = (bn0+alsq2n*exp2a)*rI*rI;

        alsq2n *= alsq2;
        real bn2 = (3*bn1+alsq2n*exp2a)*rI*rI;

        // compute the error function scaled and unscaled terms

        //RealOpenMM scale3 = getAndScaleInverseRs(particleI, particleJ, r, true, 3, TDD);
        //RealOpenMM scale5 = getAndScaleInverseRs(particleI, particleJ, r, true, 5, TDD);
        real damp      = POW(atom1.damp*atom2.damp, 1.0f/6.0f); // AA in MBPol

        bool do_scaling = (damp != 0.0) & ( damp > -50.0 ); // damp or not

        real ratio       = POW(r/damp, 4); // rA4 in MBPol

        // FIXME identify if we need to use TDDOH and so on
        int tdd = TDD;
        if ((isSameWater) && (atom1.atomType != 2) && (atom2.atomType != 2)) {
            if ((atom1.atomType == 0) | (atom2.atomType == 0)) { // one is oxygen
                tdd = TDDOH;
            } else { // both hydrogens
                tdd = TDDHH;
            }
        }
        real pgamma = thole[tdd];
        real dampForExp = -1 * pgamma * ratio;

        real scale3 = 1.0;
        if (do_scaling)
            scale3 -= EXP(dampForExp);
        real scale5 = scale3;
        if (do_scaling)
            scale5 -= (4./3.) * pgamma * EXP(dampForExp) * ratio;

        real r3 = (r*r2);
        real r5 = (r3*r2);
        real rr3 = (1-scale3)/r3;
        real rr5 = 3*(1-scale5)/r5;

        scale1 = rr3 - bn1;
        scale2 = bn2 - rr5;
    }
    else {
        scale1 = 0;
        scale2 = 0;
    }
    real dDotDelta = scale2*dot(deltaR, atom2.inducedDipole);
    atom1.field += scale1*atom2.inducedDipole + dDotDelta*deltaR;
    dDotDelta = scale2*dot(deltaR, atom2.inducedDipolePolar);
    atom1.fieldPolar += scale1*atom2.inducedDipolePolar + dDotDelta*deltaR;
    dDotDelta = scale2*dot(deltaR, atom1.inducedDipole);
    atom2.field += scale1*atom1.inducedDipole + dDotDelta*deltaR;
    dDotDelta = scale2*dot(deltaR, atom1.inducedDipolePolar);
    atom2.fieldPolar += scale1*atom1.inducedDipolePolar + dDotDelta*deltaR;
}
#else
__device__ void computeOneInteraction(AtomData& atom1, AtomData& atom2, real3 deltaR, bool isSelfInteraction) {
    if (isSelfInteraction)
        return;
    // FIXME thole copy in unique location
    const enum TholeIndices { TCC, TCD, TDD, TDDOH, TDDHH };
    const float thole[5] =  { 0.4, 0.4, 0.4,   0.4,   0.4 };

    // RealOpenMM scale3 = getAndScaleInverseRs( particleI, particleJ, r, false, 3, TDD);
    // RealOpenMM scale5 = getAndScaleInverseRs( particleI, particleJ, r, false, 5, TDD);

    real rI = RSQRT(dot(deltaR, deltaR));
    real r = RECIP(rI);
    real r2I = rI*rI;
    real rr3 = -rI*r2I;
    real rr5 = -3*rr3*r2I;

    real damp      = pow(atom1.damp*atom2.damp, 1.0f/6.0f); // AA in MBPol

    bool do_scaling = (damp != 0.0) & ( damp > -50.0 ); // damp or not

    real ratio       = pow(r/damp, 4); // rA4 in MBPol

    int tdd = TDD;
    bool isSameWater = atom1.moleculeIndex == atom2.moleculeIndex;
    if ((isSameWater) && (atom1.atomType != 2) && (atom2.atomType != 2)) {
        if ((atom1.atomType == 0) | (atom2.atomType == 0)) { // one is oxygen
            tdd = TDDOH;
        } else { // both hydrogens
            tdd = TDDHH;
        }
    }
    real pgamma = thole[TDD];
    real dampForExp = -1 * pgamma * ratio;

    real rr3_factor = 1.0;
    if (do_scaling)
        rr3_factor -= EXP(dampForExp);
    rr3 *= rr3_factor;

    real rr5_factor = rr3_factor;
    if (do_scaling)
        rr5_factor -= (4./3.) * pgamma * EXP(dampForExp) * ratio;
    rr5 *= rr5_factor;

    real dDotDelta = rr5*dot(deltaR, atom2.inducedDipole);
    atom1.field += rr3*atom2.inducedDipole + dDotDelta*deltaR;

    dDotDelta = rr5*dot(deltaR, atom2.inducedDipolePolar);
    atom1.fieldPolar += rr3*atom2.inducedDipolePolar + dDotDelta*deltaR;
    dDotDelta = rr5*dot(deltaR, atom1.inducedDipole);
    atom2.field += rr3*atom1.inducedDipole + dDotDelta*deltaR;
    dDotDelta = rr5*dot(deltaR, atom1.inducedDipolePolar);
    atom2.fieldPolar += rr3*atom1.inducedDipolePolar + dDotDelta*deltaR;
}
#endif

/**
 * Compute the mutual induced field.
 */
extern "C" __global__ void computeInducedField(
        unsigned long long* __restrict__ field, unsigned long long* __restrict__ fieldPolar, const real4* __restrict__ posq, const ushort2* __restrict__ exclusionTiles, 
        const real* __restrict__ inducedDipole, const real* __restrict__ inducedDipolePolar, unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize,
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter, const unsigned int* __restrict__ interactingAtoms,
#endif
        const float* __restrict__ damping, const int* __restrict__ moleculeIndex, const int* __restrict__ atomType) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        AtomData data;
        zeroAtomData(data);
        unsigned int atom1 = x*TILE_SIZE + tgx;
        loadAtomData(data, atom1, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);
        if (x == y) {
            // This tile is on the diagonal.

            localData[threadIdx.x].pos = data.pos;
            localData[threadIdx.x].inducedDipole = data.inducedDipole;
            localData[threadIdx.x].inducedDipolePolar = data.inducedDipolePolar;
            localData[threadIdx.x].damp = data.damp;
            localData[threadIdx.x].moleculeIndex = data.moleculeIndex;
            localData[threadIdx.x].atomType = data.atomType;

            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                real3 delta = localData[tbx+j].pos-data.pos;
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = y*TILE_SIZE+j;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS)
                    computeOneInteraction(data, localData[tbx+j], delta, atom1 == atom2);
            }
        }
        else {
            // This is an off-diagonal tile.

            loadAtomData(localData[threadIdx.x], y*TILE_SIZE+tgx, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);

            zeroAtomData(localData[threadIdx.x]);
            unsigned int tj = tgx;
            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                real3 delta = localData[tbx+tj].pos-data.pos;
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = y*TILE_SIZE+j;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS)
                    computeOneInteraction(data, localData[tbx+tj], delta, false);
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
        }

        // Write results.

        unsigned int offset = x*TILE_SIZE + tgx;
        atomicAdd(&field[offset], static_cast<unsigned long long>((long long) (data.field.x*0x100000000)));
        atomicAdd(&field[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.y*0x100000000)));
        atomicAdd(&field[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.z*0x100000000)));
        atomicAdd(&fieldPolar[offset], static_cast<unsigned long long>((long long) (data.fieldPolar.x*0x100000000)));
        atomicAdd(&fieldPolar[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.fieldPolar.y*0x100000000)));
        atomicAdd(&fieldPolar[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.fieldPolar.z*0x100000000)));

        if (x != y) {
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&field[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.x*0x100000000)));
            atomicAdd(&field[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.y*0x100000000)));
            atomicAdd(&field[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.z*0x100000000)));
            atomicAdd(&fieldPolar[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.x*0x100000000)));
            atomicAdd(&fieldPolar[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.y*0x100000000)));
            atomicAdd(&fieldPolar[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.z*0x100000000)));

        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    int pos = (int) (numTiles > maxTiles ? startTileIndex+warp*(long long)numTileIndices/totalWarps : warp*(long long)numTiles/totalWarps);
    int end = (int) (numTiles > maxTiles ? startTileIndex+(warp+1)*(long long)numTileIndices/totalWarps : (warp+1)*(long long)numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = (int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    int end = (int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        bool includeTile = true;

        // Extract the coordinates of this tile.
        
        int x, y;
#ifdef USE_CUTOFF
        if (numTiles <= maxTiles)
            x = tiles[pos];
        else
#endif
        {
            y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                y += (x < y ? -1 : 1);
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            }

            // Skip over tiles that have exclusions, since they were already processed.

            while (skipTiles[tbx+TILE_SIZE-1] < pos) {
                if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                    ushort2 tile = exclusionTiles[skipBase+tgx];
                    skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
                }
                else
                    skipTiles[threadIdx.x] = end;
                skipBase += TILE_SIZE;            
                currentSkipIndex = tbx;
            }
            while (skipTiles[currentSkipIndex] < pos)
                currentSkipIndex++;
            includeTile = (skipTiles[currentSkipIndex] != pos);
        }
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            AtomData data;
            zeroAtomData(data);

            loadAtomData(data, atom1, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);

#ifdef USE_CUTOFF
            unsigned int j = (numTiles <= maxTiles ? interactingAtoms[pos*TILE_SIZE+tgx] : y*TILE_SIZE + tgx);
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;

            loadAtomData(localData[threadIdx.x], j, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);

            zeroAtomData(localData[threadIdx.x]);

            // Compute the full set of interactions in this tile.

            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                real3 delta = localData[tbx+tj].pos-data.pos;
#ifdef USE_PERIODIC
                APPLY_PERIODIC_TO_DELTA(delta)
#endif
                int atom2 = atomIndices[tbx+tj];
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS)
                    computeOneInteraction(data, localData[tbx+tj], delta, false);
                tj = (tj + 1) & (TILE_SIZE - 1);
            }

            // Write results.

            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&field[offset], static_cast<unsigned long long>((long long) (data.field.x*0x100000000)));
            atomicAdd(&field[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.y*0x100000000)));
            atomicAdd(&field[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.field.z*0x100000000)));
            atomicAdd(&fieldPolar[offset], static_cast<unsigned long long>((long long) (data.fieldPolar.x*0x100000000)));
            atomicAdd(&fieldPolar[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.fieldPolar.y*0x100000000)));
            atomicAdd(&fieldPolar[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.fieldPolar.z*0x100000000)));

#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
            atomicAdd(&field[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.x*0x100000000)));
            atomicAdd(&field[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.y*0x100000000)));
            atomicAdd(&field[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].field.z*0x100000000)));
            atomicAdd(&fieldPolar[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.x*0x100000000)));
            atomicAdd(&fieldPolar[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.y*0x100000000)));
            atomicAdd(&fieldPolar[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].fieldPolar.z*0x100000000)));

        }
        pos++;
    }
}

extern "C" __global__ void recordInducedDipolesForDIIS(const long long* __restrict__ fixedField, const long long* __restrict__ fixedFieldPolar,
        const long long* __restrict__ fixedFieldS, const long long* __restrict__ inducedField, const long long* __restrict__ inducedFieldPolar,
        const real* __restrict__ inducedDipole, const real* __restrict__ inducedDipolePolar, const float* __restrict__ polarizability, float2* __restrict__ errors,
        real* __restrict__ prevDipoles, real* __restrict__ prevDipolesPolar, real* __restrict__ prevErrors, int iteration, bool recordPrevErrors, real* __restrict__ matrix) {
    extern __shared__ real2 buffer[];
#ifdef USE_EWALD
    const real ewaldScale = (4/(real) 3)*(EWALD_ALPHA*EWALD_ALPHA*EWALD_ALPHA)/SQRT_PI;
#else
    const real ewaldScale = 0;
#endif
    const real fieldScale = 1/(real) 0x100000000;
    real sumErrors = 0;
    real sumPolarErrors = 0;
    for (int atom = blockIdx.x*blockDim.x + threadIdx.x; atom < NUM_ATOMS; atom += blockDim.x*gridDim.x) {
        real scale = polarizability[atom];
        for (int component = 0; component < 3; component++) {
            int dipoleIndex = 3*atom+component;
            int fieldIndex = atom+component*PADDED_NUM_ATOMS;
            if (iteration >= MAX_PREV_DIIS_DIPOLES) {
                // We have filled up the buffer for previous dipoles, so shift them all over by one.
                
                for (int i = 1; i < MAX_PREV_DIIS_DIPOLES; i++) {
                    int index1 = dipoleIndex+(i-1)*NUM_ATOMS*3;
                    int index2 = dipoleIndex+i*NUM_ATOMS*3;
                    prevDipoles[index1] = prevDipoles[index2];
                    prevDipolesPolar[index1] = prevDipolesPolar[index2];
                    if (recordPrevErrors)
                        prevErrors[index1] = prevErrors[index2];
                }
            }
            
            // Compute the new dipole, and record it along with the error.
            
            real oldDipole = inducedDipole[dipoleIndex];
            real oldDipolePolar = inducedDipolePolar[dipoleIndex];
            long long fixedS = (fixedFieldS == NULL ? (long long) 0 : fixedFieldS[fieldIndex]);
            real newDipole = scale*((fixedField[fieldIndex]+fixedS+inducedField[fieldIndex])*fieldScale+ewaldScale*oldDipole);
            real newDipolePolar = scale*((fixedFieldPolar[fieldIndex]+fixedS+inducedFieldPolar[fieldIndex])*fieldScale+ewaldScale*oldDipolePolar);
            int storePrevIndex = dipoleIndex+min(iteration, MAX_PREV_DIIS_DIPOLES-1)*NUM_ATOMS*3;
            prevDipoles[storePrevIndex] = newDipole;
            prevDipolesPolar[storePrevIndex] = newDipolePolar;
            if (recordPrevErrors)
                prevErrors[storePrevIndex] = newDipole-oldDipole;
            sumErrors += (newDipole-oldDipole)*(newDipole-oldDipole);
            sumPolarErrors += (newDipolePolar-oldDipolePolar)*(newDipolePolar-oldDipolePolar);
        }
    }
    
    // Sum the errors over threads and store the total for this block.
    
    buffer[threadIdx.x] = make_real2(sumErrors, sumPolarErrors);
    __syncthreads();
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        if (threadIdx.x+offset < blockDim.x && (threadIdx.x&(2*offset-1)) == 0) {
            buffer[threadIdx.x].x += buffer[threadIdx.x+offset].x;
            buffer[threadIdx.x].y += buffer[threadIdx.x+offset].y;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
        errors[blockIdx.x] = make_float2((float) buffer[0].x, (float) buffer[0].y);
    
    if (iteration >= MAX_PREV_DIIS_DIPOLES && recordPrevErrors && blockIdx.x == 0) {
        // Shift over the existing matrix elements.
        
        for (int i = 0; i < MAX_PREV_DIIS_DIPOLES-1; i++) {
            if (threadIdx.x < MAX_PREV_DIIS_DIPOLES-1)
                matrix[threadIdx.x+i*MAX_PREV_DIIS_DIPOLES] = matrix[(threadIdx.x+1)+(i+1)*MAX_PREV_DIIS_DIPOLES];
            __syncthreads();
        }
    }
}

extern "C" __global__ void computeDIISMatrix(real* __restrict__ prevErrors, int iteration, real* __restrict__ matrix) {
    extern __shared__ real sumBuffer[];
    int j = min(iteration, MAX_PREV_DIIS_DIPOLES-1);
    for (int i = blockIdx.x; i <= j; i += gridDim.x) {
        // All the threads in this thread block work together to compute a single matrix element.

        real sum = 0;
        for (int index = threadIdx.x; index < NUM_ATOMS*3; index += blockDim.x)
            sum += prevErrors[index+i*NUM_ATOMS*3]*prevErrors[index+j*NUM_ATOMS*3];
        sumBuffer[threadIdx.x] = sum;
        __syncthreads();
        for (int offset = 1; offset < blockDim.x; offset *= 2) { 
            if (threadIdx.x+offset < blockDim.x && (threadIdx.x&(2*offset-1)) == 0)
                sumBuffer[threadIdx.x] += sumBuffer[threadIdx.x+offset];
            __syncthreads();
        }
        if (threadIdx.x == 0) {
            matrix[i+MAX_PREV_DIIS_DIPOLES*j] = sumBuffer[0];
            if (i != j)
                matrix[j+MAX_PREV_DIIS_DIPOLES*i] = sumBuffer[0];
        }
    }
}

extern "C" __global__ void updateInducedFieldByDIIS(real* __restrict__ inducedDipole, real* __restrict__ inducedDipolePolar, 
        const real* __restrict__ prevDipoles, const real* __restrict__ prevDipolesPolar, const float* __restrict__ coefficients, int numPrev) {
    for (int index = blockIdx.x*blockDim.x + threadIdx.x; index < 3*NUM_ATOMS; index += blockDim.x*gridDim.x) {
        real sum = 0;
        real sumPolar = 0;
        for (int i = 0; i < numPrev; i++) {
            sum += coefficients[i]*prevDipoles[i*3*NUM_ATOMS+index];
            sumPolar += coefficients[i]*prevDipolesPolar[i*3*NUM_ATOMS+index];
        }
        inducedDipole[index] = sum;
        inducedDipolePolar[index] = sumPolar;
    }
}

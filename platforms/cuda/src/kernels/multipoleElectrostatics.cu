#include "hip/hip_runtime.h"
#define WARPS_PER_GROUP (THREAD_BLOCK_SIZE/TILE_SIZE)

typedef struct {
    real4 posq;
    real3 force, dipole, inducedDipole, inducedDipolePolar;
    real potential;
    float damp;
    int moleculeIndex;
    int atomType;
} AtomData;

__device__ void computeOneInteractionF1(AtomData& atom1, volatile AtomData& atom2, float dScale, float pScale, float mScale, real& energy, real3& outputForce, real2& potential);

inline __device__ void loadAtomData(AtomData& data, int atom, const real4* __restrict__ posq,
const real* __restrict__ inducedDipole, const real* __restrict__ inducedDipolePolar, const float* __restrict__ damping, const int* __restrict__ moleculeIndex, const int* __restrict__ atomType) {
    data.posq = posq[atom];

    data.inducedDipole.x = inducedDipole[atom*3];
    data.inducedDipole.y = inducedDipole[atom*3+1];
    data.inducedDipole.z = inducedDipole[atom*3+2];
    data.inducedDipolePolar.x = inducedDipolePolar[atom*3];
    data.inducedDipolePolar.y = inducedDipolePolar[atom*3+1];
    data.inducedDipolePolar.z = inducedDipolePolar[atom*3+2];
    data.damp = damping[atom];
    data.moleculeIndex = moleculeIndex[atom];
    data.atomType = atomType[atom];
}

__device__ real computeDScaleFactor(unsigned int polarizationGroup, int index) {
    return (polarizationGroup & 1<<index ? 0 : 1);
}

__device__ float computeMScaleFactor(uint2 covalent, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
    return (x ? (y ? 0.0f : 0.4f) : (y ? 0.8f : 1.0f));
}

__device__ float computePScaleFactor(uint2 covalent, unsigned int polarizationGroup, int index) {
    int mask = 1<<index;
    bool x = (covalent.x & mask);
    bool y = (covalent.y & mask);
    bool p = (polarizationGroup & mask);
    return (x && y ? 0.0f : (x && p ? 0.5f : 1.0f));
}

/**
 * Compute electrostatic interactions.
 */
extern "C" __global__ void computeElectrostatics(
        unsigned long long* __restrict__ forceBuffers, unsigned long long* __restrict__ potentialBuffers, real* __restrict__ energyBuffer,
        const real4* __restrict__ posq, const uint2* __restrict__ covalentFlags, const unsigned int* __restrict__ polarizationGroupFlags,
        const ushort2* __restrict__ exclusionTiles, unsigned int startTileIndex, unsigned int numTileIndices,
#ifdef USE_CUTOFF
        const int* __restrict__ tiles, const unsigned int* __restrict__ interactionCount, real4 periodicBoxSize, real4 invPeriodicBoxSize,
        real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ, unsigned int maxTiles, const real4* __restrict__ blockCenter,
        const unsigned int* __restrict__ interactingAtoms,
#endif
        const real* __restrict__ inducedDipole,
        const real* __restrict__ inducedDipolePolar, const float* __restrict__ damping, const int* __restrict__ moleculeIndex, const int* __restrict__ atomType) {
    const unsigned int totalWarps = (blockDim.x*gridDim.x)/TILE_SIZE;
    const unsigned int warp = (blockIdx.x*blockDim.x+threadIdx.x)/TILE_SIZE;
    const unsigned int tgx = threadIdx.x & (TILE_SIZE-1);
    const unsigned int tbx = threadIdx.x - tgx;
    real energy = 0;
    __shared__ AtomData localData[THREAD_BLOCK_SIZE];
    

    // First loop: process tiles that contain exclusions.
    
    const unsigned int firstExclusionTile = FIRST_EXCLUSION_TILE+warp*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    const unsigned int lastExclusionTile = FIRST_EXCLUSION_TILE+(warp+1)*(LAST_EXCLUSION_TILE-FIRST_EXCLUSION_TILE)/totalWarps;
    for (int pos = firstExclusionTile; pos < lastExclusionTile; pos++) {
        const ushort2 tileIndices = exclusionTiles[pos];
        const unsigned int x = tileIndices.x;
        const unsigned int y = tileIndices.y;
        AtomData data;
        unsigned int atom1 = x*TILE_SIZE + tgx;
        loadAtomData(data, atom1, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);
        data.force = make_real3(0);
        data.potential = 0;
        uint2 covalent = covalentFlags[pos*TILE_SIZE+tgx];
        unsigned int polarizationGroup = polarizationGroupFlags[pos*TILE_SIZE+tgx];
        if (x == y) {
            // This tile is on the diagonal.

            localData[threadIdx.x].posq = data.posq;
            localData[threadIdx.x].dipole = data.dipole;

            localData[threadIdx.x].inducedDipole = data.inducedDipole;
            localData[threadIdx.x].inducedDipolePolar = data.inducedDipolePolar;
            localData[threadIdx.x].damp = data.damp;
            localData[threadIdx.x].moleculeIndex = data.moleculeIndex;
            localData[threadIdx.x].atomType = data.atomType;

            // Compute forces.

            for (unsigned int j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+j;
                if (atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    real2 tempPotential;
                    float d = 1.;
                    float p = 1.;
                    float m = 1.;
                    computeOneInteractionF1(data, localData[tbx+j], d, p, m, tempEnergy, tempForce, tempPotential);
                    data.force += tempForce;
                    data.potential += tempPotential.x; // FIXME divide by 2??
                    energy += 0.5f*tempEnergy;
                }
            }
            atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[atom1+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[atom1+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            atomicAdd(&potentialBuffers[atom1], static_cast<unsigned long long>((long long) (data.potential*0x100000000)));

        }
        else {
            // This is an off-diagonal tile.

            unsigned int j = y*TILE_SIZE + tgx;
            loadAtomData(localData[threadIdx.x], j, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);
            localData[threadIdx.x].force = make_real3(0);
            localData[threadIdx.x].potential = 0;
            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = y*TILE_SIZE+tj;
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    real2 tempPotential;
                    float d = 1.;
                    float p = 1.;
                    float m = 1.;
                    computeOneInteractionF1(data, localData[tbx+tj], d, p, m, tempEnergy, tempForce, tempPotential);
                    data.force += tempForce;
                    data.potential += tempPotential.x;
                    localData[tbx+tj].force -= tempForce;
                    localData[tbx+tj].potential += tempPotential.y;
                    energy += tempEnergy;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            atomicAdd(&potentialBuffers[offset], static_cast<unsigned long long>((long long) (data.potential*0x100000000)));
            offset = y*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
            atomicAdd(&potentialBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].potential*0x100000000)));

        }
    }

    // Second loop: tiles without exclusions, either from the neighbor list (with cutoff) or just enumerating all
    // of them (no cutoff).

#ifdef USE_CUTOFF
    const unsigned int numTiles = interactionCount[0];
    int pos = (int) (numTiles > maxTiles ? startTileIndex+warp*(long long)numTileIndices/totalWarps : warp*(long long)numTiles/totalWarps);
    int end = (int) (numTiles > maxTiles ? startTileIndex+(warp+1)*(long long)numTileIndices/totalWarps : (warp+1)*(long long)numTiles/totalWarps);
#else
    const unsigned int numTiles = numTileIndices;
    int pos = (int) (startTileIndex+warp*(long long)numTiles/totalWarps);
    int end = (int) (startTileIndex+(warp+1)*(long long)numTiles/totalWarps);
#endif
    int skipBase = 0;
    int currentSkipIndex = tbx;
    __shared__ int atomIndices[THREAD_BLOCK_SIZE];
    __shared__ volatile int skipTiles[THREAD_BLOCK_SIZE];
    skipTiles[threadIdx.x] = -1;
    
    while (pos < end) {
        bool includeTile = true;

        // Extract the coordinates of this tile.
        
        int x, y;
#ifdef USE_CUTOFF
        if (numTiles <= maxTiles)
            x = tiles[pos];
        else
#endif
        {
            y = (int) floor(NUM_BLOCKS+0.5f-SQRT((NUM_BLOCKS+0.5f)*(NUM_BLOCKS+0.5f)-2*pos));
            x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            if (x < y || x >= NUM_BLOCKS) { // Occasionally happens due to roundoff error.
                y += (x < y ? -1 : 1);
                x = (pos-y*NUM_BLOCKS+y*(y+1)/2);
            }

            // Skip over tiles that have exclusions, since they were already processed.

            while (skipTiles[tbx+TILE_SIZE-1] < pos) {
                if (skipBase+tgx < NUM_TILES_WITH_EXCLUSIONS) {
                    ushort2 tile = exclusionTiles[skipBase+tgx];
                    skipTiles[threadIdx.x] = tile.x + tile.y*NUM_BLOCKS - tile.y*(tile.y+1)/2;
                }
                else
                    skipTiles[threadIdx.x] = end;
                skipBase += TILE_SIZE;            
                currentSkipIndex = tbx;
            }
            while (skipTiles[currentSkipIndex] < pos)
                currentSkipIndex++;
            includeTile = (skipTiles[currentSkipIndex] != pos);
        }
        if (includeTile) {
            unsigned int atom1 = x*TILE_SIZE + tgx;

            // Load atom data for this tile.

            AtomData data;
            loadAtomData(data, atom1, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);
            data.force = make_real3(0);
            data.potential = 0;
#ifdef USE_CUTOFF
            unsigned int j = (numTiles <= maxTiles ? interactingAtoms[pos*TILE_SIZE+tgx] : y*TILE_SIZE + tgx);
#else
            unsigned int j = y*TILE_SIZE + tgx;
#endif
            atomIndices[threadIdx.x] = j;
            loadAtomData(localData[threadIdx.x], j, posq, inducedDipole, inducedDipolePolar, damping, moleculeIndex, atomType);
            localData[threadIdx.x].force = make_real3(0);
            localData[threadIdx.x].potential = 0;

            // Compute forces.

            unsigned int tj = tgx;
            for (j = 0; j < TILE_SIZE; j++) {
                int atom2 = atomIndices[tbx+tj];
                if (atom1 < NUM_ATOMS && atom2 < NUM_ATOMS) {
                    real3 tempForce;
                    real tempEnergy;
                    real2 tempPotential;
                    computeOneInteractionF1(data, localData[tbx+tj], 1, 1, 1, tempEnergy, tempForce, tempPotential);
                    data.force += tempForce;
                    data.potential += tempPotential.x;
                    localData[tbx+tj].force -= tempForce;
                    localData[tbx+tj].potential += tempPotential.y;
                    energy += tempEnergy;
                }
                tj = (tj + 1) & (TILE_SIZE - 1);
            }
            unsigned int offset = x*TILE_SIZE + tgx;
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (data.force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (data.force.z*0x100000000)));
            atomicAdd(&potentialBuffers[offset], static_cast<unsigned long long>((long long) (data.potential*0x100000000)));
#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
            atomicAdd(&forceBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.x*0x100000000)));
            atomicAdd(&forceBuffers[offset+PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.y*0x100000000)));
            atomicAdd(&forceBuffers[offset+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (localData[threadIdx.x].force.z*0x100000000)));
            atomicAdd(&potentialBuffers[offset], static_cast<unsigned long long>((long long) (localData[threadIdx.x].potential*0x100000000)));

#ifdef USE_CUTOFF
            offset = atomIndices[threadIdx.x];
#else
            offset = y*TILE_SIZE + tgx;
#endif
        }
        pos++;
    }
    energyBuffer[blockIdx.x*blockDim.x+threadIdx.x] += energy;
    
    //printf("energy = %d\n", energy);
   	//printf("%lf\n", energyBuffer[blockIdx.x*blockDim.x+threadIdx.x]);
}
